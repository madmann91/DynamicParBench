#include <vector>
#include <numeric>
#include <hip/hip_runtime_api.h>
#include "common.cuh"

namespace cuda {
    static int* nitems_per_cell;
    static int* cell_data;
    static int* start_index;
    static int* output;
    static int num_cells;
    static int num_items;
}

void process_naive  (const int* nitems_per_cell, const int* cell_data, const int* start_index, int* output, int num_cells, int num_items);
void process_blocked(const int* nitems_per_cell, const int* cell_data, const int* start_index, int* output, int num_cells, int num_items);
void process_bsearch(const int* nitems_per_cell, const int* cell_data, const int* start_index, int* output, int num_cells, int num_items);
void process_dynamic(const int* nitems_per_cell, const int* cell_data, const int* start_index, int* output, int num_cells, int num_items);

void init(const int* nitems_per_cell, const int* cell_data, int n) {
    cuda::num_cells = n;
    std::vector<int> start_index(n + 1);
    start_index[0] = 0;
    std::partial_sum(nitems_per_cell, nitems_per_cell + n, start_index.data() + 1);
    cuda::num_items = start_index.back();

    CHECK_CUDA_CALL(hipMalloc(&cuda::nitems_per_cell, sizeof(int) * n));
    CHECK_CUDA_CALL(hipMalloc(&cuda::cell_data,       sizeof(int) * n));
    CHECK_CUDA_CALL(hipMalloc(&cuda::start_index,     sizeof(int) * n));
    CHECK_CUDA_CALL(hipMalloc(&cuda::output,          sizeof(int) * cuda::num_items));

    CHECK_CUDA_CALL(hipMemcpy(cuda::nitems_per_cell, nitems_per_cell,    sizeof(int) * n, hipMemcpyHostToDevice));
    CHECK_CUDA_CALL(hipMemcpy(cuda::cell_data,       cell_data,          sizeof(int) * n, hipMemcpyHostToDevice));
    CHECK_CUDA_CALL(hipMemcpy(cuda::start_index,     start_index.data(), sizeof(int) * n, hipMemcpyHostToDevice));
}

float bench(int num_iters) {
    hipEvent_t start, end;
    CHECK_CUDA_CALL(hipEventCreate(&start));
    CHECK_CUDA_CALL(hipEventCreate(&end));

    float total = 0;
    for (int i = 0; i < num_iters; i++) {
        CHECK_CUDA_CALL(hipEventRecord(start));

        BENCH_FN(cuda::nitems_per_cell, cuda::cell_data, cuda::start_index, cuda::output, cuda::num_cells, cuda::num_items);

        CHECK_CUDA_CALL(hipEventRecord(end));
        CHECK_CUDA_CALL(hipEventSynchronize(end));
        float ms;
        CHECK_CUDA_CALL(hipEventElapsedTime(&ms, start, end));
        total += ms;
    }

    CHECK_CUDA_CALL(hipEventDestroy(start));
    CHECK_CUDA_CALL(hipEventDestroy(end));

    CHECK_CUDA_CALL(hipFree(cuda::nitems_per_cell));
    CHECK_CUDA_CALL(hipFree(cuda::cell_data));
    CHECK_CUDA_CALL(hipFree(cuda::start_index));
    return total;
}

void copy(std::vector<int>& output) {
    output.resize(cuda::num_items);
    CHECK_CUDA_CALL(hipMemcpy(output.data(), cuda::output, sizeof(int) * cuda::num_items, hipMemcpyDeviceToHost));
    CHECK_CUDA_CALL(hipFree(cuda::output));
}
